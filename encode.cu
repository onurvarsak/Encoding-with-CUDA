#include <sys/time.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

/* using gpu find decode */
__global__ void gpu_find(char** encode, char** decode){
	int index = blockIdx.x * 25 + threadIdx.x;
	if((*encode)[index] == ',')
		(*decode)[index + 1] = (*encode)[index + 1];
}

/* reading encode.txt fike */
void read_file(FILE* file, char** encode){
	char ch;
	int size = 0;
	while((ch = fgetc(file)) != EOF){
		if(ch == '\n' || ch == '\r\n' || ch == '\r')
			continue;
		(*encode)[size++] = ch;
	}
}

/* using cpu find decode */
void cpu_find(char** encode, char** decode, int encode_size){
	int i = 0;
	int decode_size = 0;
	for(i = 0; i < encode_size; i++)
		if((*encode)[i] == ',')
			(*decode)[decode_size++] = (*encode)[i + 1];
}

/* write decode to decode.txt file */
void write_file(FILE* write, char** decode, int decode_size){
	int i = 0;
	for(i = 0; i < decode_size; i++)
		fprintf(write, "%c", (*decode)[i]);
}

int main(int argc, char *argv[]) {
	struct timeval cpu_stop, cpu_start, gpu_stop, gpu_start;
	float cpu_elapsed, gpu_elapsed;
	
	FILE *read	= fopen("encodedfile.txt", "r");
	FILE *write = fopen("decode.txt", "w");
	
	int encode_size = 15360 * 100;
	int decode_size = 15360 * 4;
	
	/* CPU start */ 
	
	char* cpu_encode = (char *)malloc(sizeof(char) * encode_size);
	char* cpu_decode = (char *)malloc(sizeof(char) * decode_size);
	char* gpu_out 	 = (char *)malloc(sizeof(char) * encode_size);
	
	read_file(read, &cpu_encode);
	gettimeofday(&cpu_start, NULL);
	cpu_find(&cpu_encode, &cpu_decode, encode_size);
	gettimeofday(&cpu_stop, NULL);
	
	/* CPU end */
	
	
	
	/* GPU start */
	
	char* gpu_encode;
	char* gpu_decode;
	
	hipDeviceReset();
	
	hipMalloc((void **)&gpu_encode, (sizeof(char) * encode_size));
	hipMalloc((void **)&gpu_decode, (sizeof(char) * encode_size));
	hipMemcpy(gpu_encode, cpu_encode,(sizeof(char) * encode_size), hipMemcpyHostToDevice);
	gettimeofday(&gpu_start, NULL);
	gpu_find<<<15360 * 4, 25>>>(&gpu_encode, &gpu_decode);
	gettimeofday(&gpu_stop, NULL);
	hipMemcpy(gpu_out, gpu_decode, (sizeof(char) * encode_size), hipMemcpyDeviceToHost);
	
	/* GPU end */ 
	
	cpu_elapsed = (cpu_stop.tv_sec- cpu_start.tv_sec) * 1000.0f + (cpu_stop.tv_usec - cpu_start.tv_usec) / 1000.0f;
	gpu_elapsed = (gpu_stop.tv_sec- gpu_start.tv_sec) * 1000.0f + (gpu_stop.tv_usec - gpu_start.tv_usec) / 1000.0f;
	
	printf("CPU Code executed in %f milliseconds.\n", cpu_elapsed);
	printf("GPU Code executed in %f milliseconds.\n", gpu_elapsed);
	write_file(write, &cpu_decode, decode_size);
	
}
